#include "hip/hip_runtime.h"
/**
 * @file OptixTracerModules.cu
 * @author Ryan P. Daly (rdaly@herzog.com)
 * @brief Modules source for producing OPTIX-IR and PTX
 * @version 0.1
 * @date 2023-04-10
 * 
 * @copyright Copyright (c) 2023
 * 
 */

#include <optix.h>
#include <optix_stubs.h>

//#include "optixTriangle.h"
#include "OptixTracer.hpp"
#include <cuda/helpers.h>

#include <sutil/vec_math.h>

extern "C" {
__constant__ lidarshooter::OptixTracer::Params params;
}

extern "C" __global__ void __raygen__rg()
{
    // Lookup our location within the launch grid
    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();

    // Map our launch idx to a screen location and create a ray from the camera
    // location through the screen
    unsigned int linearIndex = idx.z * dim.y * dim.x + idx.y * dim.x + idx.x;

    // No need to do any work if we have no ray
    if (linearIndex > params.numberOfRays)
        return;

    // Trace the ray against our scene hierarchy
    unsigned int tmin;
    optixTrace(
        params.handle,
        params.rays[linearIndex].origin,
        params.rays[linearIndex].direction,
        0.0f,
        1e16f,
        0.0f,
        OptixVisibilityMask(255),
        OPTIX_RAY_FLAG_NONE,
        0,
        1,
        0,
        tmin
    );

    // Set the output tmin to the same index
    lidarshooter::Hit newHit;
    newHit.t = __uint_as_float(tmin);
    newHit.normal.x = params.rays[linearIndex].direction.x * newHit.t;
    newHit.normal.y = params.rays[linearIndex].direction.y * newHit.t;
    newHit.normal.z = params.rays[linearIndex].direction.z * newHit.t;
    params.hits[linearIndex] = newHit;
}

extern "C" __global__ void __miss__ms()
{

}

extern "C" __global__ void __closesthit__ch()
{
    // In CH program, Tmin is returned by optixGetRayTmax
    const float tmin = optixGetRayTmax();

    optixSetPayload_0(__float_as_uint(tmin));
}
